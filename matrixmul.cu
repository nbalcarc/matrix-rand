#include <stdio.h>
#include <hip/hip_runtime.h>
 

__global__ void vecmul(float *A, float* B, float *C, int size) {
    // Row and Column indexes: 
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    // Are they bellow the maximum?
    if (col < size && row < size) {
       float result = 0;
       for(int ix=0;ix<size;ix++) {
          result += A[row*size+ix]*B[ix*size+col];
       }
       C[row*size+col] = result;
    }
}


// Runs one connection, taking a value from a source node, multiplying it, and adding it to a destination node
__global__ void connection(float* mult, uint32_t* source, uint32_t* dest, float* output, uint32_t offset) {
    int id = threadIdx.x;
    atomicAdd(&output[dest[offset + id]], mult[offset + id] * output[source[offset + id]]);
}


// Normalizes a node's value so it can be used as a source node
__global__ void normalize(float* output, uint32_t offset) {
    int id = threadIdx.x;
    output[offset + id] = tanh(output[offset + id]);
}




extern "C" {
    // Verify that CUDA is available for use.
    uint32_t verify_cuda() {
        int devices = 0; 
        hipError_t err = hipGetDeviceCount(&devices); 
        return devices > 0 && err == hipSuccess;
    }

    void maxmul(float *A, float* B, float *C, int size) {

        int total = size*size;

        // Allocate device memory:
        float* gpu_A;
        float* gpu_B;
        float* gpu_C;
        int msize = total * sizeof(float);
        hipMalloc((void**)&gpu_A, msize);
        hipMemcpy(gpu_A,A,msize,hipMemcpyHostToDevice);
        hipMalloc((void**)&gpu_B, msize);
        hipMemcpy(gpu_B,B,msize,hipMemcpyHostToDevice);
        hipMalloc((void**)&gpu_C,msize);

        // Blocks & grids:
        dim3 blocks(size,size);
        dim3 grid(1,1);

        // Call the kernel:
        vecmul<<<grid,blocks>>>(gpu_A,gpu_B,gpu_C,size);

        // Get the result Matrix:
        hipMemcpy(C,gpu_C,msize,hipMemcpyDeviceToHost);

        C[0] = 2.3;

        //Free device matrices
        hipFree(gpu_A);
        hipFree(gpu_B);
        hipFree(gpu_C);
    }


    // Calculates the output of the given neural network arrays
    void calculate(
        float* mult,
        uint32_t* source,
        uint32_t* dest,
        float* output,
        uint32_t* mult_threads,
        uint32_t* output_threads,

        uint32_t connections_size,
        uint32_t output_size,
        uint32_t threads_size
    ) {

        // allocate arrays onto vram
        float* g_mult;
        uint32_t* g_source;
        uint32_t* g_dest;
        float* g_output;
        hipMalloc((void**)&g_mult, connections_size);
        hipMemcpy(g_mult, mult, connections_size, hipMemcpyHostToDevice);
        hipMalloc((void**)&g_source, connections_size);
        hipMemcpy(g_source, source, connections_size, hipMemcpyHostToDevice);
        hipMalloc((void**)&g_dest, connections_size);
        hipMemcpy(g_dest, dest, connections_size, hipMemcpyHostToDevice);
        hipMalloc((void**)&g_output, output_size);
        hipMemcpy(g_output, output, output_size, hipMemcpyHostToDevice);

        uint32_t mult_offset = 0;
        uint32_t output_offset = 0;

        // for every layer
        for (int i = 0; i < (int)threads_size; i++) {
            connection<<<1, mult_threads[i]>>>(g_mult, g_source, g_dest, g_output, mult_offset);
            mult_offset += mult_threads[i];
            normalize<<<1, output_threads[i]>>>(g_output, output_offset);
            output_offset += output_threads[i];
        }

    }

    /*
        mult: *const f32,
        source: *const usize,
        dest: *const usize,
        output: *mut f32,
        mult_threads: *const usize,
        output_threads: *const usize,

   */
}


